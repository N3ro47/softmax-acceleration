#include <stdio.h>
#include <float.h>
#include <stdint.h>
#include <vector>
#include <hip/hip_runtime.h>

static inline __device__ float atomicMax(float *addr, float value) {
    int *addr_as_int = (int *)addr;
    int old = *addr_as_int;
    int assumed;

    do {
        assumed = old;
        if (__int_as_float(assumed) >= value)
            break;
        old = atomicCAS(addr_as_int, assumed, __float_as_int(value));
    } while (assumed != old);

    return __int_as_float(old);
}

__global__ void _sum_vec(float* vec, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size) {
        vec[i] = vec[i] + vec[i + size];
    }
}

float sum_vec(std::vector<float>& vec) {
    size_t size = vec.size();

    // Ensure size is power of 2 for reduction
    size_t original_size = size;
    size_t padded_size = 1;
    while (padded_size < size) {
        padded_size <<= 1;
    }

    if (padded_size > size) {
        vec.resize(padded_size, 0.0f);
    }

    float* d_vec;
    hipMalloc(&d_vec, padded_size * sizeof(float));
    hipMemcpy(d_vec, vec.data(), padded_size * sizeof(float), hipMemcpyHostToDevice);

    size_t current_size = padded_size / 2;
    int blockSize = 256;

    while (current_size > 0) {
        int numBlocks = (current_size + blockSize - 1) / blockSize;
        _sum_vec<<<numBlocks, blockSize>>>(d_vec, current_size);
        hipDeviceSynchronize();
        current_size >>= 1;
    }

    float result;
    hipMemcpy(&result, d_vec, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_vec);

    // Restore original size if we padded it
    if (padded_size > original_size) {
        vec.resize(original_size);
    }

    return result;
}

__global__ void get_max_val(float* vec, float* max_out, size_t size) {
    extern __shared__ float shared_max[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    float local_max = -FLT_MAX;

    for (; i < size; i += blockDim.x * gridDim.x) {
        float val = vec[i];
        if (val > local_max) {
            local_max = val;
        }
    }

    shared_max[tid] = local_max;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (shared_max[tid + s] > shared_max[tid]) {
                shared_max[tid] = shared_max[tid + s];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicMax(max_out, shared_max[0]);
    }
}

__global__ void calc_exp(float* dst, float* src, float max_val, size_t size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        dst[i] = expf(src[i] - max_val);
    }
}

__global__ void calc_divis(float* dst, float sum, size_t size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        dst[i] = dst[i] / sum;
    }
}

void softmax_gpu(std::vector<float>& vec) {
    if (vec.empty()) return;

    size_t size = vec.size();
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    float *d_vec, *d_exp, *d_max;
    hipMalloc(&d_vec, size * sizeof(float));
    hipMalloc(&d_exp, size * sizeof(float));
    hipMalloc(&d_max, sizeof(float));

    hipMemcpy(d_vec, vec.data(), size * sizeof(float), hipMemcpyHostToDevice);

    float initial_max = -FLT_MAX;
    hipMemcpy(d_max, &initial_max, sizeof(float), hipMemcpyHostToDevice);

    get_max_val<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(d_vec, d_max, size);
    hipMemcpy(&initial_max, d_max, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    calc_exp<<<numBlocks, blockSize>>>(d_exp, d_vec, initial_max, size);
    hipDeviceSynchronize();

    std::vector<float> exp_vec(size);
    hipMemcpy(exp_vec.data(), d_exp, size * sizeof(float), hipMemcpyDeviceToHost);

    float sum = sum_vec(exp_vec);

    float* d_sum;
    hipMalloc(&d_sum, sizeof(float));

    calc_divis<<<numBlocks, blockSize>>>(d_exp, sum, size);
    hipDeviceSynchronize();

    hipMemcpy(vec.data(), d_exp, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_vec);
    hipFree(d_exp);
    hipFree(d_max);
    hipFree(d_sum);
}
